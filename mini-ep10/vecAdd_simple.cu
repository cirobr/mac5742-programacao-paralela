#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void vecAdd(float *A, float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char **argv) {
    //int maxBlockSize = 1024;
    int numElements = 50000;
    int size = numElements * sizeof(float);

    printf("Alocando vetores no host\n");
    float *h_A = (float *) malloc(size);
    float *h_B = (float *) malloc(size);
    float *h_C = (float *) malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Falha em alocar vetores no host\n");
        exit(-1);
    }

    printf("Inicializando vetores no host\n");
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float) (RAND_MAX);
        h_B[i] = rand() / (float) (RAND_MAX);
    }

    float *d_A;
    float *d_B;
    float *d_C;

    printf("Alocando vetores no device\n");

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    printf("Copiando memória do host para o device\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // número de threads a serem lançadas na GPU
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("Lançando um kernel com %d blocks e %d threads por bloco\n",blocksPerGrid,threadsPerBlock);

    dim3 cudaBlockSize(threadsPerBlock,1,1);
    dim3 cudaGridSize(blocksPerGrid,1,1);

    hipError_t err = hipSuccess;
    vecAdd<<<cudaGridSize, cudaBlockSize>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err !=hipSuccess){
        fprintf(stderr,
        "Failed to lauch vectorAdd kernel (error code %s)!\n",
        hipGetErrorString(err));
        exit(-1);
    }

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    printf("Fim\n");

    return 0;
}
