/*
mini EP 10

NOME: Ciro B Rosa
NUSP: 2320769
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

long getMS() {
	struct timespec s;
	clock_gettime(CLOCK_REALTIME, &s);
	return s.tv_sec*1000 + s.tv_nsec/1000000;
}

// number of tests
#define NTESTS 1 //10
#define SEED 123456
#define SIZE 1024

/*
int maxBlockSize = 1024;
int numElements  = SIZE * SIZE;
int size = numElements * sizeof(int);
int numBlocks    = (numElements + maxBlockSize - 1) / maxBlockSize;
*/

long seqSum(int *, int *);
long cudaSum(int *, int *);

int main() {
	srand(SEED);
	long seqTime = 0;
	long cudaTime = 0;

	int * reference = (int *)malloc(sizeof(int)*SIZE*SIZE);
	int result;

	for(int i = 0; i < NTESTS; i++) {
		for(int j = 0; j < SIZE*SIZE; j++) reference[j] = rand()%1024;
		seqTime += seqSum(reference, &result);
		cudaTime += cudaSum(reference, &result);
	}

	free(reference);

	printf("Average seqTime %ldms\nAvarage cudaTime %ldms\n", seqTime/NTESTS, cudaTime/NTESTS);
}

long seqSum(int *refs, int *res) {
	long t0 = getMS();

	int sum = 0;
	for(int i = 0; i < SIZE*SIZE; i++)
		sum += refs[i];
	*res = sum;

	long tf = getMS();
	return tf-t0;
}

// good tutorial link https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/

__global__ void cudaSumGPU(int *ints) {

	// you code goes here
	int numElements = SIZE * SIZE;
	int thread = blockDim.x * blockIdx.x + threadIdx.x;
	//printf("lançou thread %d\n", thread);

	int sum = 0;
	int len = numElements / blockDim.x;

	/*
	printf("blockdim %d * blockid %d + threadid %d = thread %d\n", blockDim.x,
																   blockIdx.x, 
																   threadIdx.x,
 																   thread);
	*/

	for (int i = 0; i < len; i++){
		int ind = thread * len + i;
		sum += ints[ind];
		//sum += 1;
		//printf("processou elemento %d\n", ind);
		}

	printf("soma thread %d = %d\n", thread, sum);

	//sum = 1;   //teste para ver se a soma está sendo escrita no lugar certo
	int k = numElements + threadIdx.x;
	ints[k] = sum;

	return;
}

long cudaSum(int *refs, int *res) {
	int *cudaRefs;
	int results[SIZE];

	// usem os ulimos 1024 ints da memoria para guardar o resultado
	hipMalloc(&cudaRefs, sizeof(int)*SIZE*(SIZE+1));
	hipMemcpy(cudaRefs, refs, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);

	long t0 = getMS();

	// Experiment here
	//cudaSumGPU<<< 1, SIZE >>>(cudaRefs);

	int numBlocks = 2;
	int blockSize = SIZE / numBlocks;
	cudaSumGPU<<< numBlocks, blockSize >>>(cudaRefs);

	hipMemcpy(results, cudaRefs+(SIZE*SIZE), sizeof(int)*SIZE, hipMemcpyDeviceToHost);

	int sum = 0;
	for(int i = 0; i < SIZE; i++) {
		sum += results[i];
	}
	long tf = getMS();

	hipFree(cudaRefs);

	printf("res %d   soma %d\n", *res, sum);
	puts((*res == sum)?"OK":"Err, sum mismatch");

	return tf-t0;
}
