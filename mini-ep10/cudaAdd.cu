/*
mini EP 10

NOME: Ciro B Rosa
NUSP: 2320769
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

long getMS() {
	struct timespec s;
	clock_gettime(CLOCK_REALTIME, &s);
	return s.tv_sec*1000 + s.tv_nsec/1000000;
}

// number of tests
#define NTESTS 1   //10
#define SEED 123456
#define SIZE 16   //1024

int maxBlockSize = 1024;
int numElements  = SIZE * SIZE;
int numBlocks    = (numElements + maxBlockSize - 1) / maxBlockSize;

long seqSum(int *, int *);
long cudaSum(int *, int *);

int main() {
	srand(SEED);
	long seqTime = 0;
	long cudaTime = 0;

	int * reference = (int *)malloc(sizeof(int)*SIZE*SIZE);
	int result;

	for(int i = 0; i < NTESTS; i++) {
		for(int j = 0; j < SIZE*SIZE; j++) reference[j] = rand()%1024;
		seqTime += seqSum(reference, &result);
		cudaTime += cudaSum(reference, &result);
	}

	free(reference);

	printf("Average seqTime %ldms\nAvarage cudaTime %ldms\n", seqTime/NTESTS, cudaTime/NTESTS);
}

long seqSum(int *refs, int *res) {
	long t0 = getMS();

	int sum = 0;
	for(int i = 0; i < SIZE*SIZE; i++)
		sum += refs[i];
	*res = sum;

	long tf = getMS();
	return tf-t0;
}

// good tutorial link https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/

__global__ void cudaSumGPU(int *ints) {
	// you code goes here
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	printf("i %d   blockdim %d   blockid %d   threadid %d\n", i, blockDim.x, blockIdx.x, threadIdx.x);
	
	int sum = 0;
	for(int j=0; j < SIZE*SIZE; j++){
		sum = sum + ints[j];
	}

	int k = SIZE * SIZE + threadIdx.x;
	ints[k] = sum;

	return;
}

long cudaSum(int *refs, int *res) {
	int *cudaRefs;
	int results[SIZE];

	// usem os ulimos 1024 ints da memoria para guardar o resultado
	hipMalloc(&cudaRefs, sizeof(int)*SIZE*(SIZE+1));
	hipMemcpy(cudaRefs, refs, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);

	long t0 = getMS();

	// Experiment here
	cudaSumGPU<<< 1, SIZE >>>(cudaRefs);

	hipMemcpy(results, cudaRefs+(SIZE*SIZE), sizeof(int)*SIZE, hipMemcpyDeviceToHost);

	int sum = 0;
	for(int i = 0; i < SIZE; i++) {
		sum += results[i];
	}
	long tf = getMS();

	hipFree(cudaRefs);

	printf("res %d   soma %d\n", *res, sum);
	puts((*res == sum)?"OK":"Err, sum mismatch");

	return tf-t0;
}
