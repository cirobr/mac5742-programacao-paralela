/*
mini EP 10

NOME: Ciro B Rosa
NUSP: 2320769
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

long getMS() {
	struct timespec s;
	clock_gettime(CLOCK_REALTIME, &s);
	return s.tv_sec*1000 + s.tv_nsec/1000000;
}

// number of tests
#define NTESTS 10
#define SEED 123456
#define SIZE 1024

long seqSum(int *, int *);
long cudaSum(int *, int *);

int main() {
	srand(SEED);
	long seqTime = 0;
	long cudaTime = 0;

	int * reference = (int *)malloc(sizeof(int)*SIZE*SIZE);
	int result;

	for(int i = 0; i < NTESTS; i++) {
		for(int j = 0; j < SIZE*SIZE; j++) reference[j] = rand()%1024;
		seqTime += seqSum(reference, &result);
		cudaTime += cudaSum(reference, &result);
	}

	free(reference);

	printf("Average seqTime %ldms\nAvarage cudaTime %ldms\n", seqTime/NTESTS, cudaTime/NTESTS);
}

long seqSum(int *refs, int *res) {
	long t0 = getMS();

	int sum = 0;
	for(int i = 0; i < SIZE*SIZE; i++)
		sum += refs[i];
	*res = sum;

	long tf = getMS();
	return tf-t0;
}

// good tutorial link https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/

__global__ void cudaSumGPU(int *ints) {

	// you code goes here
	int numElements = SIZE * SIZE;
	int thread = blockDim.x * blockIdx.x + threadIdx.x;

	int sum = 0;
	int len = numElements / blockDim.x;
	for (int i = 0; i < len; i++){
		int ind = thread * len + i;
		sum += ints[ind];
		}

	int k = numElements + threadIdx.x;
	ints[k] = sum;

	return;
}

long cudaSum(int *refs, int *res) {
	int *cudaRefs;
	int results[SIZE];

	// usem os ulimos 1024 ints da memoria para guardar o resultado
	hipMalloc(&cudaRefs, sizeof(int)*SIZE*(SIZE+1));
	hipMemcpy(cudaRefs, refs, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);

	long t0 = getMS();

	// Experiment here
	int numBlocks = 1;
	int blockSize = SIZE / numBlocks;
	cudaSumGPU<<< numBlocks, blockSize >>>(cudaRefs);

	hipMemcpy(results, cudaRefs+(SIZE*SIZE), sizeof(int)*SIZE, hipMemcpyDeviceToHost);

	int sum = 0;
	for(int i = 0; i < SIZE; i++) {
		sum += results[i];
	}
	long tf = getMS();

	hipFree(cudaRefs);

	printf("res %d   soma %d\n", *res, sum);
	puts((*res == sum)?"OK":"Err, sum mismatch");

	return tf-t0;
}
